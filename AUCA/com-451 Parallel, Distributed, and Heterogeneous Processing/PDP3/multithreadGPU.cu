#include "hip/hip_runtime.h"
#include "multithreadGPU.h"
#include <stdio.h>

unsigned long size_Points = NUMBER_OF_POINTS * sizeof(Points);
unsigned long size_Parameters = sizeof(Parameters);


// GPU_Palette openPallete_Points() {
  
//   unsigned long theSize = NUMBER_OF_POINTS * 3;
//   unsigned long memSize = theSize * sizeof(float);

//   float* xmap = (float*) malloc(memSize);
//   float* ymap = (float*) malloc(memSize);
//   float* zmap = (float*) malloc(memSize);

//   for(int i = 0; i < theSize; i++){
//     xmap[i] = getRandNum();
//     ymap[i] = getRandNum();
//     zmap[i] = getRandNum();
//   }

//   GPU_Palette P2 = initGPUPalette(NUMBER_OF_POINTS, 1);

//   hipMemcpy(P2.red,    xmap, memSize, cH2D);
//   hipMemcpy(P2.green,  ymap, memSize, cH2D);
//   hipMemcpy(P2.blue,   zmap, memSize, cH2D);

//   free(xmap);
//   free(ymap);
//   free(zmap);
// }

// int updatePoint(GPU_Palette* P) {
//   calculatePoint <<<P->gBlocks, P->gThreads>>> (P->red, P->green, P->blue);  
// }

// __global__ void calculatePoint(float* xP, float* yP, float* zP) {

//   int x = threadIdx.x + (blockIdx.x * blockDim.x);
//   int y = threadIdx.y + (blockIdx.y * blockDim.y);
//   int vecIdx = x + (y * blockDim.x * gridDim.x);

//   double delta_x, delta_y, delta_z;
  
//   delta_x = t * (10.0 * (yP[vecIdx] - xP[vecIdx]));
//   delta_y = t * ( (xP[vecIdx] * (28.0 - xP[vecIdx])) - yP[vecIdx]);
//   delta_z = t * ( (xP[vecIdx] * yP[vecIdx]) - (2.666 * xP[vecIdx]) );

//   xP[vecIdx] += delta_x;
//   yP[vecIdx] += delta_y;
//   zP[vecIdx] += delta_z;

// //   static float minX = -20.0;
// //   static float maxX = 20.0;
// //   static float minY = -30.0;
// //   static float maxY = 30.0;

// //   static float xRange = fabs(maxX - minX);
// //   static float xScalar = 0.9 * (gWIDTH/xRange);

// //   static float yRange = fabs(maxY - minY);
// //   static float yScalar = 0.9 * (gHEIGHT/yRange);

// //   Points->points[tid].xIdx = round(xScalar * (Points->points[tid].x - minX));
// //   Points->points[tid].yIdx = round(yScalar * (Points->points[tid].y - minY));

//   Points->points[tid].xIdx = floor((Points->points[tid].x * 32) + 960); // (X * scalar) + (gWidth/2)
//   Points->points[tid].yIdx = floor((Points->points[tid].y * 18) + 540); // (Y * scalar) + (gHeight/2)
// }


Points initPointMem_GPU(Points* points) {

  Points *PointsGPU;

  hipMalloc((void**) &PointsGPU, size_Points);  
  hipMemcpy(PointsGPU, &points, size_Points, cH2D);
  
  printf("%f\n", points->points[0].start_x);
  printf("%f\n", PointsGPU->points[0].start_x);
  
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) {
    printf("\nCUDA Error: %s\n", hipGetErrorString(err));
    exit(-1);
  }

  return *PointsGPU;
}

Parameters initParametersMem_GPU(Parameters* parameters) {

  Parameters *ParametersGPU;

  hipMalloc((void**) &ParametersGPU, size_Parameters);  
  hipMemcpy(ParametersGPU, &parameters, size_Parameters, cH2D);
  
  printf("%f\n", parameters->b);
  printf("%f\n", &(ParametersGPU->b));
  
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) {
    printf("\nCUDA Error: %s\n", hipGetErrorString(err));
    exit(-1);
  }

  return *ParametersGPU;
}

int updatePoint_GPU(Points* PointsGPU, Parameters* ParametersGPU) {
  calculatePoint <<<NUMBER_OF_POINTS, 1>>> (PointsGPU, ParametersGPU);
  return 0;
}

int shareFromGPUtoHost(Points* points, Points* Points_GPU) {
  hipMemcpy(points, Points_GPU, size_Points, cD2H);

  return 0;
}

int freePointMem_GPU(Points* Points_GPU, Parameters* ParametersGPU) {
  hipFree(&Points_GPU);
  hipFree(&ParametersGPU);

  return 0;
}

int initStartingPoints_GPU(Points* Points_GPU) {
  initStartingPoint <<<1, NUMBER_OF_POINTS>>> (Points_GPU);

  return 0;
}

__global__ void initStartingPoint(Points* Points) {

  int tid = threadIdx.x;
  printf("tid: %d\n", tid);

  Points->points[tid].start_x = 0.5;
  Points->points[tid].start_y = 0.5;
  Points->points[tid].start_z = 0.5;

  Points->points[tid].x = Points->points[tid].start_x*10;
  Points->points[tid].y = Points->points[tid].start_y*10;
  Points->points[tid].z = Points->points[tid].start_z*10;

  Points->points[tid].red = Points->points[tid].start_x;
  Points->points[tid].green = Points->points[tid].start_y;
  Points->points[tid].blue = Points->points[tid].start_z;

  if((Points->points[tid].red >= Points->points[tid].green) && (Points->points[tid].red >= Points->points[tid].blue))
    Points->points[tid].color_heatTransfer = 0;
  else if (Points->points[tid].green >= Points->points[tid].blue)
    Points->points[tid].color_heatTransfer = 1;
  else
    Points->points[tid].color_heatTransfer = 2;
}

__global__ void calculatePoint(Points* PointsGPU, Parameters* ParametersGPU) {

  int tid = blockIdx.x;
  printf("gpu_mt_tid: %d\n", tid);
  // printf("%f\n", (&gpu_t)->points->points[tid].start_x);

//   gpu_t->points->points[tid].delta_x = t * (gpu_t->parameters.a * (gpu_t->points->points[tid].y - gpu_t->points->points[tid].x));
//   gpu_t->points->points[tid].delta_y = t * ( (gpu_t->points->points[tid].x * (gpu_t->parameters.b - gpu_t->points->points[tid].z)) - gpu_t->points->points[tid].y);
//   gpu_t->points->points[tid].delta_z = t * ( (gpu_t->points->points[tid].x * gpu_t->points->points[tid].y) - (gpu_t->parameters.c * gpu_t->points->points[tid].z) );

//   gpu_t->points->points[tid].x += gpu_t->points->points[tid].delta_x;
//   gpu_t->points->points[tid].y += gpu_t->points->points[tid].delta_y;
//   gpu_t->points->points[tid].z += gpu_t->points->points[tid].delta_z;

//   printf("%d: %f\n", tid, gpu_t->points->points[tid].x);

// //   static float minX = -20.0;
// //   static float maxX = 20.0;
// //   static float minY = -30.0;
// //   static float maxY = 30.0;

// //   static float xRange = fabs(maxX - minX);
// //   static float xScalar = 0.9 * (gWIDTH/xRange);

// //   static float yRange = fabs(maxY - minY);
// //   static float yScalar = 0.9 * (gHEIGHT/yRange);

// //   gpu_t->points->points[tid].xIdx = round(xScalar * (gpu_t->points->points[tid].x - minX));
// //   gpu_t->points->points[tid].yIdx = round(yScalar * (gpu_t->points->points[tid].y - minY));

//   gpu_t->points->points[tid].xIdx = floor((gpu_t->points->points[tid].x * 32) + 960); // (X * scalar) + (gWidth/2)
//   gpu_t->points->points[tid].yIdx = floor((gpu_t->points->points[tid].y * 18) + 540); // (Y * scalar) + (gHeight/2)
}