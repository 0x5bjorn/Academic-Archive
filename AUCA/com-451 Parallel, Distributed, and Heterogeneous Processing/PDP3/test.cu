
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

typedef struct
{
    float a, b;
} point;

__global__ void testKernel(point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].a = 1.1;
    p[i].b = 2.2;
}

int main(void)
{
        // set number of points 
    int numPoints    = 16,
        gpuBlockSize = 4,
        pointSize    = sizeof(point),
        numBytes     = numPoints * pointSize,
        gpuGridSize  = numPoints / gpuBlockSize;

        // allocate memory
    point *cpuPointArray,
          *gpuPointArray;
    cpuPointArray = (point*)malloc(numBytes);
    hipMalloc((void**)&gpuPointArray, numBytes);

        // launch kernel
    testKernel<<<gpuGridSize,gpuBlockSize>>>(gpuPointArray);

        // retrieve the results
    hipMemcpy(cpuPointArray, gpuPointArray, numBytes, hipMemcpyDeviceToHost);
    printf("testKernel results:\n");
    for(int i = 0; i < numPoints; ++i)
    {
        printf("point.a: %f, point.b: %f\n",cpuPointArray[i].a,cpuPointArray[i].b);
    }
 
    printf("point.a: %f, point.b: %f\n", (gpuPointArray)->a, (gpuPointArray)->b);
 
        // deallocate memory
    free(cpuPointArray);
    hipFree(gpuPointArray);

    return 0;
}